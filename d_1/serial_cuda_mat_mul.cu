#include <math.h>
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;


__global__
void mat_mul(float *result, float *mat1, float *mat2, int rows1, int cols1, int cols2)
{

    for (int i=0; i < rows1; i++)
    {
        for (int j = 0; j < cols2; j++)
        {
            float count = 0;
            for (int k=0; k < cols1; k++)
            {
                count += mat1[i * cols1 + k]*mat2[k*cols2 + j];
            } 
        result[i*cols2 + j] = count;
        }
    }
    
}

int main()
{
    int a = 64;  // rows of first matrix
    int b = 64;  // cols of first matrix / rows of second matrix
    int c = 64;  // cols of second matrix

    // Host arrays
    float h_m1[a][b];
    float h_m2[b][c];
    float h_res[a][c];

    // Device pointers
    float *d_m1, *d_m2, *d_res;

    // Allocate device memory
    hipMalloc(&d_m1, a * b * sizeof(float));
    hipMalloc(&d_m2, b * c * sizeof(float));
    hipMalloc(&d_res, a * c * sizeof(float));

    // Initialize host arrays
    for (int i = 0; i < a; i++) {
        for (int j = 0; j < b; j++) {
            h_m1[i][j] = i * j + 2.0f;
        }
    }

    for (int i = 0; i < b; i++) {
        for (int j = 0; j < c; j++) {
            h_m2[i][j] = i * j + 2.0f;
        }
    }

    // Copy data to device
    hipMemcpy(d_m1, h_m1, a * b * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_m2, h_m2, b * c * sizeof(float), hipMemcpyHostToDevice);

    // Set up grid and block dimensions
    int blockSize = 256;
    int numThreads = (a*b + blockSize - 1 )/blockSize;

    // Launch kernel
    mat_mul<<<blockSize, numThreads>>>(d_res, d_m1, d_m2, a, b, c);

    // Copy result back to host
    hipMemcpy(h_res, d_res, a * c * sizeof(float), hipMemcpyDeviceToHost);

    // Print a small portion of the result (for verification)
    cout << "First few elements of result matrix:" << endl;
    for (int i = 0; i < min(5, a); i++) {
        for (int j = 0; j < min(5, c); j++) {
            cout << h_res[i][j] << " ";
        }
        cout << endl;
    }

    // Free device memory
    hipFree(d_m1);
    hipFree(d_m2);
    hipFree(d_res);

    return 0;
}