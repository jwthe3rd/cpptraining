#define N 1000000



#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void vector_add(float *out, float *a, float *b, int n) {

	int index = 0;
	int stride = 256;

	for (int i = index; i<n; i+=stride) {
		out[i] = a[i] + b[i];
	}

}

int main() {


	float *a, *b, *out;
	float *d_a, *d_b, *d_out;

	// Allocate memory 
	//
	a = (float*)malloc(sizeof(float)*N);
	b = (float*)malloc(sizeof(float)*N);
	out = (float*)malloc(sizeof(float)*N);


	for (int i = 0; i<N; i++){
		a[i] = 1.0f; b[i] = 2.0f;
	}

	hipMalloc((void**)&d_a, sizeof(float)*N);
	hipMemcpy(d_a, a, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_b, sizeof(float)*N);
	hipMemcpy(d_b, b, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_out, sizeof(float)*N);
	hipMemcpy(d_out, out, sizeof(float)*N, hipMemcpyHostToDevice);

	// CUDA uses thread blocks to assign parallel compute tasks
	// <<<M, T>>>, grid of M threadblocks, T threads in block


	vector_add<<<1,256>>>(out, a, b, N);


	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);

	free(a);
	free(b);
	free(out);

	hipError_t err = hipGetLastError();  // add
if (err != hipSuccess) printf("Cuda Error %f",hipGetErrorString); // add
}
