#define N 1000000



#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void vector_add(float *out, float *a, float *b, int n) {


	// example of a grid-stride loop

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x*gridDim.x;

	for (int i = index; i<n; i+=stride) {
		out[i] = a[i] + b[i];
	}

}

int main() {

/* cpu mem alloc
	float *a = new float[N];
	float *b = new float[N];
	float *out = new float[N];
*/

	// Unified mem alloc
	//
	
	float *a, *b, *out;
	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;

	hipMallocManaged(&a, N*sizeof(float));
	hipMallocManaged(&b, N*sizeof(float));
	hipMallocManaged(&out, N*sizeof(float));

	for (int i = 0; i<N; i++){
		a[i] = 1.0f; b[i] = 2.0f;
	}

	// kernel syntax <<<M, T>>> M thread blocks of T threads
	// blocks of threads are multiples of 32

	vector_add<<<numBlocks, blockSize>>>(out, a, b, N);

	hipDeviceSynchronize(); // cpu wait til gpu done

	float maxError = 0.0f;

	for (int i = 0; i<N; i++){
		maxError = fmax(maxError, fabs(out[i]-3.0f));
	}

	std::cout << "Max Error: " << maxError << std::endl;

	/* Free mem from cpu mem
	//
	delete [] a;
	delete [] b;
	delete [] out; */

	hipFree(a);
	hipFree(b);
	hipFree(out);


	return 0;

}
