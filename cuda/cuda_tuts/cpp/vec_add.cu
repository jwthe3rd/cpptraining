#define N 1000000



#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void vector_add(float *out, float *a, float *b, int n) {

	int index = 0;
	int stride = 1;

	for (int i = index; i<n; i+=stride) {
		out[i] = a[i] + b[i];
	}

}

int main() {

/* cpu mem alloc
	float *a = new float[N];
	float *b = new float[N];
	float *out = new float[N];
*/

	// Unified mem alloc
	//
	
	float *a, *b, *out;

	hipMallocManaged(&a, N*sizeof(float));
	hipMallocManaged(&b, N*sizeof(float));
	hipMallocManaged(&out, N*sizeof(float));

	for (int i = 0; i<N; i++){
		a[i] = 1.0f; b[i] = 2.0f;
	}

	// kernel syntax <<<M, T>>> M thread blocks of T threads
	// blocks of threads are multiples of 32

	vector_add<<<1,1>>>(out, a, b, N);

	hipDeviceSynchronize(); // cpu wait til gpu done

	float maxError = 0.0f;

	for (int i = 0; i<N; i++){
		maxError = fmax(maxError, fabs(out[i]-3.0f));
	}

	std::cout << "Max Error: " << maxError << std::endl;

	/* Free mem from cpu mem
	//
	delete [] a;
	delete [] b;
	delete [] out; */

	hipFree(a);
	hipFree(b);
	hipFree(out);


	return 0;

}
