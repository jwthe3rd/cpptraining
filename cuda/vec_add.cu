#define N 1000000



#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void vector_add(float *out, float *a, float *b, int n) {

	for (int i = 0; i<n; i++) {
		out[i] = a[i] + b[i];
	}

}

int main() {


	float *a, *b, *out;
	float *d_a, *d_b, *d_out;

	// Allocate memory 
	//
	a = (float*)malloc(sizeof(float)*N);
	b = (float*)malloc(sizeof(float)*N);
	out = (float*)malloc(sizeof(float)*N);


	for (int i = 0; i<N; i++){
		a[i] = 1.0f; b[i] = 2.0f;
	}

	hipMalloc((void**)&d_a, sizeof(float)*N);
	hipMemcpy(d_a, a, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_b, sizeof(float)*N);
	hipMemcpy(d_b, b, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_out, sizeof(float)*N);
	hipMemcpy(d_out, out, sizeof(float)*N, hipMemcpyHostToDevice);

	vector_add<<<1,1>>>(out, a, b, N);


	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);

	free(a);
	free(b);
	free(out);

}
